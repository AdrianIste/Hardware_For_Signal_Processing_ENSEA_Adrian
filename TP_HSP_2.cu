#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


dim3 thread_dim (28,28);
dim3 thread_dim2 (14,14);
#define WIDTH 28
#define HEIGHT 28

float *raw_data;
float *C1_data;
float *S1_data;
float *C1_kernel;
float *Mean_kernel;

float *raw_datacuda;
float *C1_datacuda;
float *S1_datacuda;
float *C1_kernelcuda;
float *Mean_kernelcuda;


void MatrixInit_dim2_0_1(float *M, int n, int p) { //matrix n*p with values between 0 and 1
	
	for (int i = 0; i < p; ++i) {
		for (int t =0; t<n;t++) {
			M[i*n+t]=(rand()%101)/100.0f; //between 0 and 1 and i*n+p to index 
		}
	}
}

void MatrixInit_dim3(float *M, int m, int n, int p) { //matrix m n p initialized with 0
	
	for (int i = 0; i < p; ++i) {
		for (int t =0; t<n;t++) {
			for (int z=0; z<m;z++) {
				
				M[i*n+t+z*n*p]=0; //init to 0
			}
		}
	}
}
void MatrixInit_dim3_0_1(float *M, int m, int n, int p) {  //matrix m n p initialized with values between 0 and 1
	
	for (int i = 0; i < p; ++i) {
		for (int t =0; t<n;t++) {
			for (int z=0; z<m;z++) {
				
				M[i*n+t+z*n*p]=(rand()%101)/100.0f; //between 0 and 1 
			}
		}
	}			
}

void ArrayInit(float *M, int size, float value) { 
	for (int i = 0; i < size; ++i) {
		M[i]=value;
	}			
}

void MatrixPrint(float *M, int n, int p) {
	for (int i = 0; i < p; ++i) {
		for (int t =0; t<n;t++) {
			printf("%10.4f  ", M[i*n+t]); //f for float
		}
		printf("\n"); //f for float
	}
}
__global__ void cudaConvolve(float *K, float *I, float *out, int ni, int ki){ //pi number of lines of the images, ni number of columns, K=kernels,I=image, ki=size of the kernel
	int i = blockIdx.x; //kernel number
	int jx = threadIdx.x; //convolve position
	int jy = threadIdx.y; //convolve position

	float conv=0;
	for (int l=0; l<ki;l++) {
		for (int c=0;c<ki;c++) {
			conv+=K[i*ki*ki+l*ki+c]*I[(jx+l)*ni+c+jy];//equation just linearisation de l image.
		}
	}
	out[i * 28 * 28 + jx * 28 + jy] =conv;
}
			
		
__global__ void cudaSampling(float *K, float *I, float *out, int ni, int ki){ //pi number of lines of the images, ni number of columns, K=kernels,I=image, ki=size of the kernel
	int i = blockIdx.x; //kernel number
	int jx = threadIdx.x; //convolve position
	int jy = threadIdx.y; //convolve position

	float mean=0;
	for (int l=0; l<ki;l++) {
		for (int c=0;c<ki;c++) {
			mean+=K[l*ki+c]*I[i*ni*ni+(2*jx+l)*ni+c+2*jy];//equation just linearisation de l image.
		}
	}
	out[i * 14 * 14 + jx * 14 + jy] =mean;
}

__device__ float activation_tanh(float M) {
    return 2.0f / (1.0f + expf(-2.0f * M)) - 1.0f;
}


__global__ void apply_activation_tanh(float *data, int size) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    data[i] = activation_tanh(data[i]);  
    
}

		
	
	
int main() {
	raw_data=(float*)malloc(32*32*sizeof(float));
	C1_data=(float*)malloc(6*28*28*sizeof(float));
	S1_data=(float*)malloc(6*14*14*sizeof(float));
	C1_kernel=(float*)malloc(6*5*5*sizeof(float));
	Mean_kernel=(float*)malloc(2*2*sizeof(float));
	

	
	MatrixInit_dim2_0_1(raw_data,32,32); //matrix 32*32 initialized with values between 0 and 1
	MatrixInit_dim3(C1_data,6,28,28); //matrix 6*28*28 initialized to 0
	MatrixInit_dim3(S1_data,6,14,14); //matrix 6*28*28 initialized to 0
	MatrixInit_dim3_0_1(C1_kernel,6,5,5); //matrix 6*5*5 initialized with values between 0 and 1
	ArrayInit(Mean_kernel,2*2,0.25);//initialisation for mean sampling
	
	//to test
	ArrayInit(raw_data,32*32,0);
	for (int i=0; i<32; i++){
		for (int j=0; j<32; j++){
			if (i == j)
				raw_data[i*32+j]=1;
		}
	}
	
	ArrayInit(C1_data,6*28*28,-1);
	for (int c=0; c<6; c++){
		ArrayInit(C1_kernel+(c*5*5), 5*5, (float)(c+1)/6);
	}
	
	MatrixPrint(raw_data,32,32);
		
	
	hipMalloc((void**)&raw_datacuda, sizeof(float)*32*32);
    hipMalloc((void**)&C1_datacuda, sizeof(float)*6*28*28);
    hipMalloc((void**)&S1_datacuda, sizeof(float)*6*14*14);
    hipMalloc((void**)&C1_kernelcuda, sizeof(float)*6*5*5);
    hipMalloc((void**)&Mean_kernelcuda, sizeof(float)*2*2);
    
    hipMemcpy(raw_datacuda, raw_data, sizeof(float)*32*32, hipMemcpyHostToDevice); //Cpu to GPU
	hipMemcpy(C1_datacuda,C1_data , sizeof(float)*6*28*28, hipMemcpyHostToDevice);
	hipMemcpy(S1_datacuda, S1_data, sizeof(float)*6*14*14, hipMemcpyHostToDevice); //Cpu to GPU
	hipMemcpy(C1_kernelcuda, C1_kernel, sizeof(float)*6*5*5, hipMemcpyHostToDevice);
	hipMemcpy(Mean_kernelcuda, Mean_kernel, sizeof(float)*2*2, hipMemcpyHostToDevice); //Cpu to GPU

	cudaConvolve<<<6,thread_dim>>>(C1_kernelcuda,raw_datacuda,C1_datacuda,32,5);
	cudaSampling<<<6,thread_dim2>>>(Mean_kernelcuda,C1_datacuda,S1_datacuda,28,2);
	//activation_tanh<<<4,294>>>(S1_datacuda); //comment if needed
	
	hipMemcpy(C1_data, C1_datacuda, sizeof(float)*6*28*28, hipMemcpyDeviceToHost);
	hipMemcpy(S1_data, S1_datacuda, sizeof(float)*6*14*14, hipMemcpyDeviceToHost);
	
	

	for (int i = 0; i < 6; ++i) {
		MatrixPrint(&C1_kernel[i*5*5],5,5);
		printf("convolve \n");
		MatrixPrint(&C1_data[i*28*28],28,28);
	}
	
	for (int i = 0; i < 6; ++i) {
		MatrixPrint(Mean_kernel,2,2);
		printf("Sampling \n");
		MatrixPrint(&S1_data[i*14*14],14,14);
	}
	
	
	free(raw_data);
	free(C1_data);
	free(S1_data);
	free(C1_kernel);
	free(Mean_kernel);
	
	hipFree(raw_datacuda);
	hipFree(C1_datacuda);
	hipFree(S1_datacuda);
	hipFree(C1_kernelcuda);
	hipFree(Mean_kernelcuda);
	
	
}
	
	
