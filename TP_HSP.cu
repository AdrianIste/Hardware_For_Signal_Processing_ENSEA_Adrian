#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <time.h>
float *M1;
float *M2;
float *Mout;
float *Mout2;
float *MoutMult;
float *MoutMult2;

float *M1cuda;
float *M2cuda;
float *Moutcuda;
float *MoutMultcuda;

int n; //number of lines
int p; //number of columns

int grid_size;
int block_size;

void MatrixInit(float *M, int n, int p) { //we put ** because it works
	
	for (int i = 0; i < p; ++i) {
		for (int t =0; t<n;t++) {
			M[i*n+t]=(rand()%201-100)/100.0f; //between -1 and 1 and i*n+p to index 
		}
	}
}

void MatrixPrint(float *M, int n, int p) {
	for (int i = 0; i < p; ++i) {
		for (int t =0; t<n;t++) {
			printf("%10.4f  ", M[i*n+t]); //f for float
		}
		printf("\n"); //f for float
	}
}

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p) { //matrix addition
	for (int i = 0; i < p; ++i) {
		for (int t =0; t<n;t++) {
			Mout[i*n+t]=M1[i*n+t]+M2[i*n+t];
		}
	}
}


void MatrixMult(float *M1, float *M2, float *Mout, int n) { //matrix multiplication
	float temp;
	for (int i = 0; i < n; ++i) {
		for (int j=0; j<n;j++) {
			
			temp=0; //to compute the addition for each line and columns
			for (int t =0; t<n;t++) {
				temp+=M1[i*n+t]*M2[t*n+j];
			}
		Mout[i*n+j]=temp;
	}
	}
}
			
			
		
		
		
	
__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p){ //addition on cuda
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid<n*p) {
		Mout[tid]=M1[tid]+M2[tid];
		}
	
}
	
__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n) { //mult on cuda
	int i = blockIdx.x; //line
	int j = threadIdx.x; //column

	float temp=0; //to compute the addition for each line and columns
	for (int t =0; t<n;t++) {
		temp+=M1[i*n+t]*M2[t*n+j];
			}
		Mout[i*n+j]=temp;
}
		

		
			
			
			
int main(int argc, char *argv[]) {
	n=atoi(argv[1]);
	p=atoi(argv[2]);
	grid_size=atoi(argv[3]);
	block_size=atoi(argv[4]);
	M1=(float*)malloc(n*p*sizeof(float));
	M2=(float*)malloc(n*p*sizeof(float));
	Mout=(float*)malloc(n*p*sizeof(float));
	Mout2=(float*)malloc(n*p*sizeof(float));

	MoutMult=(float*)malloc(n*n*sizeof(float));
	MoutMult2=(float*)malloc(n*n*sizeof(float));
	
	MatrixInit(M1,n,p);
	MatrixInit(M2,n,p);
	clock_t start = clock();
	MatrixAdd(M1,M2,Mout,n,p);
	clock_t end = clock();
	
	clock_t start2 = clock();
	MatrixMult(M1,M2,MoutMult,n);
	clock_t end2 = clock();
	float elapsed_time = (float)(end - start) / CLOCKS_PER_SEC; 
	float elapsed_time2 = (float)(end2 - start2) / CLOCKS_PER_SEC; 

	
	printf("Matrice M1 :\n");
	MatrixPrint(M1,n,p);
	printf("Matrice M2 :\n");
	MatrixPrint(M2,n,p);
	printf("Matrice Mout :\n");
	MatrixPrint(Mout,n,p);
	printf("MatrixAdd Time: %f seconds\n", elapsed_time);
	printf("Matrice MoutMult :\n");
	MatrixPrint(MoutMult,n,n);
	printf("MatrixMult Time: %f seconds\n", elapsed_time2);
	hipMalloc((void**)&M1cuda, sizeof(float)*grid_size*block_size);
    hipMalloc((void**)&M2cuda, sizeof(float)*grid_size*block_size);
    hipMalloc((void**)&Moutcuda, sizeof(float)*grid_size*block_size);
    hipMalloc((void**)&MoutMultcuda, sizeof(float)*grid_size*block_size);
	

	
	hipMemcpy(M1cuda, M1, sizeof(float)*grid_size*block_size, hipMemcpyHostToDevice); //Cpu to GPU
	hipMemcpy(M2cuda, M2, sizeof(float)*grid_size*block_size, hipMemcpyHostToDevice);
	
	clock_t start3 = clock();

    cudaMatrixAdd<<<grid_size,block_size>>>(M1cuda,M2cuda,Moutcuda,grid_size,block_size);
    clock_t end3 = clock();

    clock_t start4 = clock();

    cudaMatrixMult<<<grid_size,block_size>>>(M1cuda,M2cuda,MoutMultcuda,grid_size);

   	clock_t end4 = clock();

    hipMemcpy(Mout2, Moutcuda, sizeof(float)*grid_size*block_size, hipMemcpyDeviceToHost);
    hipMemcpy(MoutMult2, MoutMultcuda, sizeof(float)*grid_size*block_size, hipMemcpyDeviceToHost);
	float elapsed_time3 = (float)(end3 - start3) / CLOCKS_PER_SEC; 
	float elapsed_time4 = (float)(end4- start4) / CLOCKS_PER_SEC; 

    
	printf("Matrice MoutAddCuda :\n");
	MatrixPrint(Mout2,grid_size,block_size);
	printf("MatrixAdd cuda Time: %f seconds\n", elapsed_time3);

	printf("Matrice MoutMultCuda :\n");
	MatrixPrint(MoutMult2,grid_size,block_size);
	printf("MatrixMult cuda Time: %f seconds\n", elapsed_time4);

	
    
    
    
	free(M1);
	free(M2);
	free(Mout);
	free(Mout2);
	free(MoutMult);
	free(MoutMult2);

	hipFree(M1cuda);
    hipFree(M2cuda);
    hipFree(Moutcuda);
    hipFree(MoutMultcuda);

	
	
	
	
	return 0;

}
	
